#include "hip/hip_runtime.h"

#include "mandelbrot_gpu.h"

__device__
auto cuda_norm(hipDoubleComplex const& c) {
    return c.x * c.x + c.y * c.y;
}

__global__ 
static void fractal_gpu_kernel(pfc::bmp::pixel_t* const pixels, size_t const width, size_t const height, hipDoubleComplex const ll, double const d) {
    auto t{ blockIdx.x * blockDim.x + threadIdx.x };

    hipDoubleComplex c{ ll.x + d * (t % width), ll.y + d * (t / width) };

    if (t < width*height)
        pixels[t] = iterate(c);

}

hipError_t fractal_gpu(dim3 big, dim3 tib, pfc::bmp::pixel_t* const pixels, size_t const width, size_t const height, hipDoubleComplex const ll, hipDoubleComplex const ur)
{

    auto const complex_width{ ur.x - ll.x };
    auto const d{ complex_width / width };

    fractal_gpu_kernel<<< big, tib >>>(pixels, width, height, ll, d);

    return hipGetLastError();
}
